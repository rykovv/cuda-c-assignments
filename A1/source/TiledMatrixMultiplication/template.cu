#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> 
#include <> 
#include <wb.h>

#define TILE_WIDTH 16 	//do not change this value

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBColumns) {
  //@@ Insert code to implement tiled matrix multiplication here
  //@@ You have to use shared memory to write this kernel
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x,  by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;

    float tmpC = 0;

    // assuming TILE_WIDTH is a multiple of numBColumns
    //          blockDim.{x,y} are multiples of numARows and numBColumns
    for (int p = 0; p < numBColumns / TILE_WIDTH; p++) {
        // ty and tx must be < TILE_WIDTH

        if ((p * TILE_WIDTH + ty < numARows) && Col < numAColumns) {
            ds_A[ty][tx] = A[(p * TILE_WIDTH + ty) * numAColumns + Col];
        } else {
            ds_A[ty][tx] = 0;
        }
        if (Row < numARows && (p * TILE_WIDTH + tx < numAColumns)) {
            ds_B[ty][tx] = B[Row * numBColumns + (p * TILE_WIDTH + tx)];
        } else {
            ds_B[ty][tx] = 0;
        }
        __syncthreads();

        if (Row < numARows && Col < numBColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                tmpC += ds_A[i][tx] * ds_B[ty][i];
            }
        }
        __syncthreads();

        if (Row < numARows && Col < numBColumns) {
            C[Row * numAColumns + Col] = tmpC;
        }
    }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
  
  hostC = NULL;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = 0;
  numCColumns = 0;
  //@@ Allocate the hostC matrix
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid((numCRows - 1) / TILE_WIDTH + 1, (numCColumns - 1) / TILE_WIDTH + 1, 1);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
