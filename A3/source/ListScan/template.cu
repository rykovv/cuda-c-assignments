#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define BLOCK_SIZE 512

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float *input, float *output, float *aux, int len) {
    //@@ Modify the body of this kernel to generate the scanned blocks
    //@@ Make sure to use the workefficient version of the parallel scan
    //@@ Also make sure to store the block sum to the aux array

    // Data loading
    __shared__ float XY[2*BLOCK_SIZE];

    int i = 2*blockIdx.x*blockDim.x + threadIdx.x;

    if (i < len) {
        XY[threadIdx.x] = input[i];
    }
    if (i + blockDim.x < len) {
        XY[threadIdx.x + blockDim.x] = input[i + blockDim.x];
    }

    // Reduction phase
    for (unsigned stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
        // once the shared memory is populated or intermediate computation is done
        __syncthreads();
        // legacy indexing
        int index = (threadIdx.x + 1)*stride*2 - 1;
        if (index < 2*BLOCK_SIZE) {
            XY[index] += XY[index - stride];
        }
    }

    // Post reduction reverse phase
    for (unsigned stride = BLOCK_SIZE/2; stride > 0; stride /= 2) {
        // once previous computations are done
        __syncthreads();
        // legacy index
        int index = (threadIdx.x + 1)*stride*2 - 1;
        if (index + stride < 2*BLOCK_SIZE) {
            XY[index + stride] += XY[index];
        }
    }
    // finish all the computations
    __syncthreads();
    // store outputs
    if (i < len) {
        output[i] = XY[threadIdx.x];
    }
    if (i + blockDim.x < len) {
        output[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
    }

    if (aux) {
        if (i == len - 1) {
            aux[blockIdx.x] = XY[threadIdx.x];
        } else // if (threadIdx.x == 2*BLOCK_SIZE - 1) {
            // what is faster BLOCK_SIZE comparisons + 1 store or BLOCK_SIZE stores?
            aux[blockIdx.x] = XY[2*BLOCK_SIZE - 1];
        }
    }
}

__global__ void addScannedBlockSums(float *output, float *aux, int len) {
	//@@ Modify the body of this kernel to add scanned block sums to
	//@@ all values of the scanned blocks
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (blockIdx.x > 0 && i < len) {
        output[i] += aux[blockIdx.x - 1];
    }
}

__host__ unsigned nblocks_scan (unsigned len) {
    return ((len - 1) / 2*BLOCK_SIZE + 1);
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output 1D list
  float *deviceInput;
  float *deviceOutput;
  float *deviceAuxArray, *deviceAuxScannedArray;
  int numElements; // number of elements in the input/output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  wbTime_start(GPU, "Allocating device memory.");
  //@@ Allocate device memory
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  //you can assume that aux array size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)
  wbCheck(hipMalloc((void **)&deviceAuxArray, nblocks_scan(numElements) * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceAuxScanArray, nblocks_scan(numElements) * sizeof(float)));
  wbTime_stop(GPU, "Allocating device memory.");

  wbTime_start(GPU, "Clearing output device memory.");
  //@@ zero out the deviceOutput using hipMemset() by uncommenting the below line
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output device memory.");

  wbTime_start(GPU, "Copying input host memory to device.");
  //@@ Copy input host memory to device
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input host memory to device.");

  //@@ Initialize the grid and block dimensions here

  dim3 DimGrid(nblocks_scan(numElements), 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  //@@ You need to launch scan kernel twice: 1) for generating scanned blocks
  //@@ (hint: pass deviceAuxArray to the aux parameter)
  //@@ and 2) for generating scanned aux array that has the scanned block sums.
  //@@ (hint: pass NULL to the aux parameter)
  //@@ Then you should call addScannedBlockSums kernel.
  //invoke CUDA kernel
  scan <<< DimGrid, DimBlock >>> (
	  deviceInput,
	  deviceOutput,
	  deviceAuxArray,
	  numElements
  );

  // reduce the number of threads to be launched
  // "you can assume that aux array size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)"
  // that implies we'll need to launch only one block, but let's be generic
  // dim3 DimGrid(nblocks_scan(nblocks_scan(numElements)), 1, 1);
  dim3 DimBlock(1, 1, 1);

  scan <<< DimGrid, DimBlock >>> (
	  deviceAuxArray,
	  deviceAuxScanArray,
    NULL,
	  nblocks_scan(numElements)
  );
  hipDeviceSynchronize();

  dim3 DimBlock(2*BLOCK_SIZE, 1, 1);
  addScannedBlockSums <<< DimGrid, DimBlock >>> (
	  deviceOutput,
	  deviceAuxScanArray,
	  numElements
  );
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy results from device to host
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Deallocate device memory
  wbCheck(hipFree(deviceInput));
  wbCheck(hipFree(deviceOutput));
  wbCheck(hipFree(deviceAuxArray));
  wbCheck(hipFree(deviceAuxScanArray));
  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
