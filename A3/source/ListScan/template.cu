#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define BLOCK_SIZE 512

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__host__ __device__ unsigned nblocks_scan(unsigned len) {
    return ((len - 1)/(2 * BLOCK_SIZE) + 1);
}

__host__ __device__ void debug(float* arr, unsigned len) {
    printf("debugging array of length %d\n", len);
    for (int i = min(len - 1, 2 * BLOCK_SIZE - 1); i < len; i += 2 * BLOCK_SIZE) {
        printf("arr[%d] = %.2f\n", i, arr[i]);
    }
    if (len % (2 * BLOCK_SIZE)) {
        printf("arr[%d] = %.2f\n", len - 1, arr[len - 1]);
    }
}

__host__ __device__ void print_array(float* arr, int arr_len) {
    printf("printing array of length %d\n", arr_len);
    for (int i = arr_len; i > 0; i--) {
        printf("arr[%d] = %.2f\n", arr_len - i, arr[arr_len - i]);
    }
}

__global__ void scan (float *input, float *output, float *aux, int len) {
    //@@ Modify the body of this kernel to generate the scanned blocks
    //@@ Make sure to use the workefficient version of the parallel scan
    //@@ Also make sure to store the block sum to the aux array

    // Data loading
    __shared__ float XY[2*BLOCK_SIZE];

    int i = 2*blockIdx.x*blockDim.x + threadIdx.x;

    if (i < len) {
        XY[threadIdx.x] = input[i];
    }
    if (i + blockDim.x < len) {
        XY[threadIdx.x + blockDim.x] = input[i + blockDim.x];
    }

    // Reduction phase
    for (unsigned stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
        // once the shared memory is populated or intermediate computation is done
        __syncthreads();
        // legacy indexing
        int index = (threadIdx.x + 1)*stride*2 - 1;
        if (index < 2*BLOCK_SIZE) {
            XY[index] += XY[index - stride];
        }
    }

    // Post reduction reverse phase
    for (unsigned stride = BLOCK_SIZE/2; stride > 0; stride /= 2) {
        // once previous computations are done
        __syncthreads();
        // legacy index
        int index = (threadIdx.x + 1)*stride*2 - 1;
        if (index + stride < 2*BLOCK_SIZE) {
            XY[index + stride] += XY[index];
        }
    }
    // finish all the computations
    __syncthreads();
    // store outputs
    if (i < len) {
        output[i] = XY[threadIdx.x];
    }
    if (i + blockDim.x < len) {
        output[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
    }

    if (aux) {
        if (i == len - 1) {
            aux[blockIdx.x] = XY[threadIdx.x];
        } else if (threadIdx.x == BLOCK_SIZE - 1) {
            // what is faster BLOCK_SIZE comparisons + 1 store or BLOCK_SIZE stores?
            aux[blockIdx.x] = XY[2*BLOCK_SIZE - 1];
        }
    }

    __syncthreads();
    if (i == len) {
        printf("\nPrinting from scan() aux %s\n", aux != NULL? "is not NULL" : "is NULL");
        if (aux) {
            printf("output array\n");
            debug(output, len);
            printf("aux array\n");
            print_array(aux, nblocks_scan(len));
        } else {
            printf("output array\n");
            print_array(output, len);
        }
    }
}

__global__ void addScannedBlockSums(float *output, float *aux, int len) {
	//@@ Modify the body of this kernel to add scanned block sums to
	//@@ all values of the scanned blocks
    int i = (blockIdx.x+1)*blockDim.x + threadIdx.x;
    //printf("addScannedBlockSums len = %d and i = %d\n", len, i);
    if (i < len) {
        //if ((!(i % 2*BLOCK_SIZE) && i < len) || ((i % 2*BLOCK_SIZE) && i == len)) {
        //    printf("output[%d]=%.2f += aux[%d]=%.2f => output[%d]=%.2f\n", i, output[i], blockIdx.x, aux[blockIdx.x], i, output[i] + aux[blockIdx.x]);
        //}
        output[i] += aux[blockIdx.x];
    }

    /*
    __syncthreads();
    if (i == len) {
        printf("\nPrinting from addScannedBlockSums()\n");
        printf("output array\n");
        debug(output, len);
        printf("aux array\n");
        print_array(aux, (len-1)/BLOCK_SIZE+1);
    }
    */
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output 1D list
  float *deviceInput;
  float *deviceOutput;
  float *deviceAuxArray, *deviceAuxScannedArray;
  int numElements; // number of elements in the input/output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numElements);

  debug(hostInput, numElements);

  wbTime_start(GPU, "Allocating device memory.");
  //@@ Allocate device memory
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  //you can assume that aux array size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)
  wbCheck(hipMalloc((void **)&deviceAuxArray, nblocks_scan(numElements) * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceAuxScannedArray, nblocks_scan(numElements) * sizeof(float)));
  wbTime_stop(GPU, "Allocating device memory.");

  wbTime_start(GPU, "Clearing output device memory.");
  //@@ zero out the deviceOutput using hipMemset() by uncommenting the below line
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  wbTime_stop(GPU, "Clearing output device memory.");

  wbTime_start(GPU, "Copying input host memory to device.");
  //@@ Copy input host memory to device
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float), hipMemcpyHostToDevice));
  wbTime_stop(GPU, "Copying input host memory to device.");

  //@@ Initialize the grid and block dimensions here

  dim3 DimGrid(nblocks_scan(numElements), 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  //@@ You need to launch scan kernel twice: 1) for generating scanned blocks
  //@@ (hint: pass deviceAuxArray to the aux parameter)
  //@@ and 2) for generating scanned aux array that has the scanned block sums.
  //@@ (hint: pass NULL to the aux parameter)
  //@@ Then you should call addScannedBlockSums kernel.
  //invoke CUDA kernel
  printf("launching scan 1st time\n");
  scan <<< DimGrid, DimBlock >>> (
	  deviceInput,
	  deviceOutput,
	  deviceAuxArray,
	  numElements
  );

  // reduce the number of threads to be launched
  // "you can assume that aux array size would not need to be more than BLOCK_SIZE*2 (i.e., 1024)"
  // that implies we'll need to launch only one block, but let's be generic
  // dim3 DimGrid(nblocks_scan(nblocks_scan(numElements)), 1, 1);
  dim3 DimBlockAux(1, 1, 1);

  printf("launching scan 2nd time\n");
  scan <<< DimGrid, DimBlockAux >>> (
	  deviceAuxArray,
	  deviceAuxScannedArray,
      NULL,
	  nblocks_scan(numElements)
  );
  hipDeviceSynchronize();

  dim3 DimGridAuxSum(nblocks_scan(numElements)-1, 1, 1);
  dim3 DimBlockAuxSum(2*BLOCK_SIZE, 1, 1);

  printf("launching last scan sum\n");
  addScannedBlockSums <<< DimGridAuxSum, DimBlockAuxSum >>> (
	  deviceOutput,
	  deviceAuxScannedArray,
	  numElements
  );
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy results from device to host
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float), hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Deallocate device memory
  wbCheck(hipFree(deviceInput));
  wbCheck(hipFree(deviceOutput));
  wbCheck(hipFree(deviceAuxArray));
  wbCheck(hipFree(deviceAuxScannedArray));
  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
