#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define NUM_BINS    4096
#define BLOCK_SIZE  512
#define SAT_MAX     127

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line,
    bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
        if (abort)
            exit(code);
    }
}

__global__ void histogram(unsigned int* input, unsigned int* bins,
    unsigned int num_elements,
    unsigned int num_bins) {
    //@@ Write the kernel that computes the histogram
    //@@ Make sure to use the privitization technique
    //(hint: since NUM_BINS=4096 is larger than maximum allowed number of threads per block,
    //be aware that threads would need to initialize more than one shared memory bin
    //and update more than one global memory bin)
    __shared__ unsigned int private_bins[NUM_BINS];

    if (threadIdx.x < NUM_BINS) {
        private_bins[threadIdx.x] = 0;
    }
    __syncthreads();

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    unsigned int nbin = 0;

    while (i < num_elements) {
        nbin = input[i] / ((num_elements - 1) / num_bins + 1);
        atomicAdd(&(private_bins[nbin]), 1);
        i += stride;
    }
    __syncthreads();

    if (threadIdx.x < NUM_BINS) {
        atomicAdd(&(bins[threadIdx.x]), private_bins[threadIdx.x]);
    }
}

__global__ void saturate(unsigned int* bins, unsigned int num_bins) {
    //@@ Write the kernel that applies saturtion to counters (i.e., if the bin value is more than 127, make it equal to 127)
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < num_bins) {
        bins[i] = min(bins[i], SAT_MAX);
    }
}

int main(int argc, char* argv[]) {
    wbArg_t args;
    int inputLength;
    unsigned int* hostInput;
    unsigned int* hostBins;
    unsigned int* deviceInput;
    unsigned int* deviceBins;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput = (unsigned int*)wbImport(wbArg_getInputFile(args, 0),
        &inputLength, "Integer");
    hostBins = (unsigned int*)malloc(NUM_BINS * sizeof(unsigned int));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);
    wbLog(TRACE, "The number of bins is ", NUM_BINS);

    wbTime_start(GPU, "Allocating device memory");
    CUDA_CHECK(hipMalloc((void**)&deviceInput, inputLength * sizeof(unsigned int)));
    CUDA_CHECK(hipMalloc((void**)&deviceBins, NUM_BINS * sizeof(unsigned int)));
    //@@ Allocate device memory here
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(GPU, "Allocating device memory");

    wbTime_start(GPU, "Copying input host memory to device");
    //@@ Copy input host memory to device
    CUDA_CHECK(hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(GPU, "Copying input host memory to device");

    wbTime_start(GPU, "Clearing the bins on device");
    //@@ zero out the deviceBins using hipMemset()
    CUDA_CHECK(hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int)));
    wbTime_stop(GPU, "Clearing the bins on device");

    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid((inputLength - 1) / BLOCK_SIZE + 1, 1, 1);
    //dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    wbLog(TRACE, "Launching kernel");
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Invoke kernels: first call histogram kernel and then call saturate kernel
    histogram << < DimGrid, DimBlock >> > (
        deviceInput,
        deviceBins,
        inputLength,
        NUM_BINS
        );

    saturate << < DimGrid, DimBlock >> > (deviceBins, NUM_BINS);
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output device memory to host");
    //@@ Copy output device memory to host
    CUDA_CHECK(hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(Copy, "Copying output device memory to host");

    wbTime_start(GPU, "Freeing device memory");
    //@@ Free the device memory here
    CUDA_CHECK(hipFree(deviceInput));
    CUDA_CHECK(hipFree(deviceBins));
    wbTime_stop(GPU, "Freeing device memory");

    wbSolution(args, hostBins, NUM_BINS);

    free(hostBins);
    free(hostInput);
    return 0;
}
