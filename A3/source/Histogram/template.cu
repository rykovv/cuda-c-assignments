#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <wb.h>

#define NUM_BINS 4096
#define BLOCK_SIZE 512 

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

__global__ void histogram(unsigned int *input, unsigned int *bins,
	unsigned int num_elements,
	unsigned int num_bins) {
	//@@ Write the kernel that computes the histogram
	//@@ Make sure to use the privitization technique
	//(hint: since NUM_BINS=4096 is larger than maximum allowed number of threads per block, 
	//be aware that threads would need to initialize more than one shared memory bin 
	//and update more than one global memory bin)
}

__global__ void saturate(unsigned int *bins, unsigned int num_bins) {
	//@@ Write the kernel that applies saturtion to counters (i.e., if the bin value is more than 127, make it equal to 127)
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (unsigned int *)wbImport(wbArg_getInputFile(args, 0),
                                       &inputLength, "Integer");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  wbTime_start(GPU, "Allocating device memory");
  //@@ Allocate device memory here
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating device memory");

  wbTime_start(GPU, "Copying input host memory to device");
  //@@ Copy input host memory to device
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input host memory to device");
	
  wbTime_start(GPU, "Clearing the bins on device");
  //@@ zero out the deviceBins using hipMemset() 
  wbTime_stop(GPU, "Clearing the bins on device");

  //@@ Initialize the grid and block dimensions here

  wbLog(TRACE, "Launching kernel");
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Invoke kernels: first call histogram kernel and then call saturate kernel
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output device memory to host");
  //@@ Copy output device memory to host
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output device memory to host");

  wbTime_start(GPU, "Freeing device memory");
  //@@ Free the device memory here
  wbTime_stop(GPU, "Freeing device memory");

  wbSolution(args, hostBins, NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
